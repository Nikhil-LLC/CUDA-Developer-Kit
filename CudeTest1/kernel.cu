#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

__global__ void kernel(int** c) {
    int index = threadIdx.x;
    printf("Thread | val: %i\n", index * 10);
    c[index][0] = index * 10;  // Access the int pointed to by c[index]
}

int main() {
    int length = 10;

    // Allocate host memory
    int* h_c = new int[length];
    
    // Allocate device memory for array of pointers
    int** dev_c;
    hipMalloc((void**)&dev_c, sizeof(int*) * length);

    // Allocate device memory for each integer and copy pointers to them
    int* temp;
    for (int i = 0; i < length; i++) {
        hipMalloc((void**)&temp, sizeof(int));
        hipMemcpy(&dev_c[i], &temp, sizeof(int*), hipMemcpyHostToDevice);
    }

    cout << "Launching kernel..." << endl;

    // Launch kernel
    kernel<<<1, length>>>(dev_c);
    hipDeviceSynchronize();

    cout << "Kernel result:" << endl;

    // Copy results from device to host
    hipMemcpy(h_c, dev_c, sizeof(int*) * length, hipMemcpyDeviceToHost);

    // Print results
    cout << "Result: ";
    for (int i = 0; i < length; i++) {
      cout << h_c[i] << " ";
    }

    // Clean up
    for (int i = 0; i < length; i++) {
        hipFree(dev_c[i]);  // Free each allocated int
    }
    hipFree(dev_c);  // Free the array of pointers
    delete[] h_c;  // Free host memory

    return 0;
}